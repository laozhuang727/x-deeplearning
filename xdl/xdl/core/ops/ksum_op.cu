#include "hip/hip_runtime.h"
/*
 * Copyright 1999-2017 Alibaba Group.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
*/

#include "xdl/core/ops/ksum_op.h"
#include "xdl/core/framework/op_registry.h"
#include "xdl/core/lib/common_defines.h"
#include "xdl/core/lib/atomic.h"
#include "xdl/core/lib/binary_search.h"
#include "xdl/core/framework/gpu/gpu_device.h"

#include <hip/hip_runtime_api.h>

namespace xdl {
namespace {

template <typename T, typename I>
__global__ void KSumKernel(const T* peb, const I* pidx, const T* pval,
                           const I* pgrp, size_t grp_size, size_t eb_dim,
                           bool average, T* pout) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= pgrp[grp_size - 1]) return;

  const I* p = LowerBound(pgrp, pgrp + grp_size, idx + 1);
  size_t grp_idx = p - pgrp;

  size_t grp_width = (grp_idx == 0) ? p[0]
                                    : (p[0] - p[-1]);
  if (grp_width == 0) return;

  const T* src = peb + pidx[idx] * eb_dim;
  T* dst = pout + grp_idx * eb_dim;
  for (size_t k = 0; k < eb_dim; ++k) {
    T val = (pval != nullptr) ? pval[idx] * src[k] : src[k];
    if (average) val /= grp_width;
    common::gpu_atomic_add<T>(val, dst + k);
  }
}

}  // namespace

template <typename T, typename I>
class KSumGpuOp : public GpuOpKernel {
 public:
  Status Init(OpKernelConstruction* ctx) override;
  Status LaunchKernel(OpKernelContext* ctx, CudaStream* stream) override;
 private:
  bool average_;
};

template <typename T, typename I>
Status KSumGpuOp<T, I>::Init(OpKernelConstruction* ctx) {
  XDL_CHECK_STATUS(ctx->GetAttr("average", &average_));
  return Status::Ok();
}

template <typename T, typename I>
Status KSumGpuOp<T, I>::LaunchKernel(OpKernelContext* ctx, CudaStream* stream) {
  Tensor embed, index, value, segment, group, output;
  XDL_CHECK_STATUS(ctx->GetInput(0, &embed));
  XDL_CHECK_COND(2 == embed.Shape().Size(),
                 Status::ArgumentError("embed input dim must be 2"));
  XDL_CHECK_STATUS(ctx->GetInput(1, &index));
  XDL_CHECK_COND(1 == index.Shape().Size(),
                 Status::ArgumentError("index input dim must be 1"));
  XDL_CHECK_STATUS(ctx->GetInput(2, &value));
  XDL_CHECK_COND(value.Shape().NumElements() == index.Shape().NumElements() ||
                 value.Shape().NumElements() == 0,
                 Status::ArgumentError("value input size must match index"));
  XDL_CHECK_STATUS(ctx->GetInput(3, &segment));
  XDL_CHECK_COND(1 == segment.Shape().Size(),
                 Status::ArgumentError("segment input dim must be 1"));
  XDL_CHECK_STATUS(ctx->GetInput(4, &group));
  XDL_CHECK_COND(1 == group.Shape().Size(),
                 Status::ArgumentError("group input dim must be 1"));
  
  T* peb = embed.Raw<T>();
  I* pidx = index.Raw<I>();
  T* pval = value.Raw<T>();
  I* pseg = segment.Raw<I>();
  I* pgrp = group.Raw<I>();

  if (value.Shape().NumElements() == 0) {
    pval = nullptr;
  }

  size_t eb_dim = embed.Shape()[1];
  size_t seg_size = segment.Shape().NumElements();
  size_t id_size = index.Shape().NumElements();
  size_t grp_size = seg_size;

  TensorShape out_shape({seg_size, eb_dim});
  if (group.Shape().NumElements() == 0) {
    pgrp = pseg;
  } else {
    grp_size = group.Shape().NumElements();
    XDL_CHECK(grp_size % seg_size == 0) << "group must be divided by segment";
    size_t grp_num = grp_size / seg_size;
    out_shape.Set(1, out_shape[1] * grp_num);
  }
  XDL_CHECK_STATUS(ctx->AllocateOutput(0, out_shape, &output));
  T* pout = output.Raw<T>();
  size_t bytes = sizeof(T) * out_shape.NumElements();
  CUDA_CHECK(hipMemsetAsync(pout, 0, bytes, stream->GetInternal()));
  if (id_size == 0) return Status::Ok();

  size_t blocks = CUDA_GET_BLOCKS(id_size);
  KSumKernel<T, I><<<
      blocks,
      CUDA_GET_THREADS(id_size, blocks),
      0,
      stream->GetInternal()>>>(peb, pidx, pval, pgrp, grp_size,
                               eb_dim, average_, pout);
  return Status::Ok();
}

#define REGISTER_GPU_KERNEL(T, I)            \
  XDL_REGISTER_KERNEL(KSum, KSumGpuOp<T, I>) \
    .Device("GPU")                           \
    .AttrDataType<T>("dtype")                \
    .AttrDataType<I>("itype")

REGISTER_GPU_KERNEL(float, int32_t);
REGISTER_GPU_KERNEL(float, int64_t);
REGISTER_GPU_KERNEL(double, int32_t);
REGISTER_GPU_KERNEL(double, int64_t);

#undef REGISTER_GPU_KERNEL

}  // namespace xdl
