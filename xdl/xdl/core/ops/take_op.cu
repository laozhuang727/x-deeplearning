#include "hip/hip_runtime.h"
/*
 * Copyright 1999-2017 Alibaba Group.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
*/

#include "xdl/core/ops/take_op.h"
#include "xdl/core/framework/op_registry.h"
#include "xdl/core/lib/common_defines.h"
#include "xdl/core/framework/gpu/gpu_device.h"

namespace xdl {
namespace {

template <typename T, typename I>
__global__ void TakeOpKernel(const T* pin,
                             const I* pind,
                             size_t col,
                             size_t num,
                             T* pout) {
  const size_t k = blockIdx.x * blockDim.x + threadIdx.x;
  if (k >= num) return;
  const size_t i = k / col, j = k % col;
  pout[k] = pin[pind[i] * col + j];
}

}  // namespace

template <typename T, typename I>
class TakeGpuOp : public GpuOpKernel {
 public:
  Status Init(OpKernelConstruction* ctx) override {
    return Status::Ok();
  }
  Status LaunchKernel(OpKernelContext* ctx, CudaStream* stream) override;
};

template <typename T, typename I>
Status TakeGpuOp<T, I>::LaunchKernel(OpKernelContext* ctx, CudaStream* stream) {
  Tensor feature, indicator, output;
  XDL_CHECK_STATUS(ctx->GetInput(0, &feature));
  XDL_CHECK_STATUS(ctx->GetInput(1, &indicator));
  XDL_CHECK_COND(1 == indicator.Shape().Size(),
                 Status::ArgumentError("indicator must be rank 1 tensor"));

  auto fea_dims = feature.Shape().Dims();
  std::vector<size_t> dims(fea_dims.begin(), fea_dims.end());
  dims[0] = indicator.Shape().NumElements();
  TensorShape out_shape(dims);
  XDL_CHECK_STATUS(ctx->AllocateOutput(0, out_shape, &output));

  size_t row = dims[0];
  size_t col = feature.Shape().NumElements() / feature.Shape()[0];
  size_t num = row * col;
  T* pin = feature.Raw<T>(), *pout = output.Raw<T>();
  I* pind = indicator.Raw<I>();

  hipStream_t st = stream->GetInternal();
  if (num == 0) {
    CUDA_CHECK(hipMemsetAsync(pout, 0, sizeof(T) * out_shape.NumElements(), st));
    return Status::Ok();
  }
  size_t blocks = CUDA_GET_BLOCKS(num);
  TakeOpKernel<T, I><<<
      blocks,
      CUDA_GET_THREADS(num, blocks),
      0,
      st>>>(pin, pind, col, num, pout);
  return Status::Ok();
}

#define REGISTER_GPU_KERNEL(T, I)              \
  XDL_REGISTER_KERNEL(TakeOp, TakeGpuOp<T, I>) \
  .Device("GPU")                               \
  .AttrDataType<T>("dtype")                    \
  .AttrDataType<I>("itype")

REGISTER_GPU_KERNEL(float, int32_t);
REGISTER_GPU_KERNEL(float, int64_t);
REGISTER_GPU_KERNEL(double, int32_t);
REGISTER_GPU_KERNEL(double, int64_t);

#undef REGISTER_GPU_KERNEL

}  // namespace xdl
